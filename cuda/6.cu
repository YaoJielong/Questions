#include "hip/hip_runtime.h"
//CUDA code corresponding to week6.cpp, applying artificial viscoity and perioidc boundary in my code

#include <iostream>
#include <cmath>
#include <fstream>
#include "tools.h"

using namespace std;

#define LATTICE_WIDTH 50 //number of particles per row in the latticle initially
#define LATTICE_HEIGHT 50 //number of particles per column in the latticle initially
#define LATTICE_SIZE 2500 //total number of particles in the particle lattice

#define NUM_BLOCKS 20 //number of launched blocks 
#define NUM_THREADS_PER_BLOCK 125 //number of threads within a block


struct Coordinates {

    int number;//number of particles in the lattice
    int dimensions;//spatial dimensions
    double timestep; //global timestep for every particle
    double x_min; //the minimal value of x coordinate of the particle lattice
    double y_min; //the minimal value of y coordinate of the particle lattice
    double xbound; // length of simulation box boundary in x direction
    double ybound; // length of simulation box boundary in y direction
    
    double rx[LATTICE_SIZE];//an array of x coordinate of particles
    double ry[LATTICE_SIZE];//an array of y coordinate of particles
    
    double vx[LATTICE_SIZE];//an array of velosity in x direction of particles
    double vy[LATTICE_SIZE];

    double dvxdt[LATTICE_SIZE];//equation (44) in https://www.sciencedirect.com/science/article/pii/S0021999110006753 in x direction
    double dvydt[LATTICE_SIZE];//equation (44) in https://www.sciencedirect.com/science/article/pii/S0021999110006753 in y direction

    double u[LATTICE_SIZE]; //internal energy u in equation (2.72) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf
    double dudt[LATTICE_SIZE]; //discretised internal energy (time derivative of u) in equation (2.74) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf

    double h[LATTICE_SIZE];//an array of smoothing lengths of particles
    double m[LATTICE_SIZE];//an array of mass of particles
    double density[LATTICE_SIZE];//an array of density of particles
    double p[LATTICE_SIZE];//pressure at each particle
    double omega[LATTICE_SIZE];//an array of omega values of particles, Ωa in equation (2.42) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf
    double temp_timestep[LATTICE_SIZE];//store temporary timestep of each particle; we need to find the minimal one from this array to be the global timestep for every particle

} coordinates;


//create a lattice of particles in square arrangement
Coordinates lattice_init_square(int nx = LATTICE_WIDTH, int ny = LATTICE_HEIGHT){

    //initialization
    coordinates.number = LATTICE_SIZE;
    coordinates.dimensions = 2;
    coordinates.timestep = 0.0;
    coordinates.x_min = 0.0; // 0 in this case
    coordinates.y_min = 0.0; // 0 in this case
    coordinates.xbound = 1.0;
    coordinates.ybound = 1.0;


    for(int i = 0; i < LATTICE_SIZE; i++){
        coordinates.vx[i] = 0.0;
        coordinates.vy[i] = 0.0;
        coordinates.dvxdt[i] = 0.0;
        coordinates.dvydt[i] = 0.0;
        coordinates.dudt[i] = 0.0;
        coordinates.density[i] = 0.0;
        coordinates.p[i] = 0.0;
        coordinates.omega[i] = 0.0;
        coordinates.temp_timestep[i] = 0.0;
        coordinates.u[i] = 1.0;
        coordinates.h[i] = 0.02; 
        coordinates.m[i] = 0.43; 
    }
    
    double x_offset = 0.5 / LATTICE_WIDTH;
    double y_offset = 0.5 / LATTICE_HEIGHT;
    int counter = 0;

    for(int i = 0; i < LATTICE_WIDTH; i++){//create particle lattice
        for(int j = 0; j < LATTICE_HEIGHT; j++){
            coordinates.rx[counter] = (double)i / LATTICE_WIDTH + x_offset;
            coordinates.ry[counter] = (double)j / LATTICE_HEIGHT + y_offset;
            counter++;
        }
    }

    return coordinates;

}


__global__ void calculate_density(Coordinates* d_out, Coordinates* d_in){// based on equation(2) in https://www.sciencedirect.com/science/article/abs/pii/S0021999110006753

    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    int lattice_size = d_in->number;
    

    d_out->number = d_in->number;
    d_out->dimensions = d_in->dimensions;
    d_out->timestep = d_in->timestep;
    d_out->x_min = d_in->x_min;
    d_out->y_min = d_in->y_min;
    d_out->xbound = d_in->xbound;
    d_out->ybound = d_in->ybound;

    d_out->rx[thread_id] = d_in->rx[thread_id];
    d_out->ry[thread_id] = d_in->ry[thread_id];

    d_out->vx[thread_id] = d_in->vx[thread_id];
    d_out->vy[thread_id] = d_in->vy[thread_id];

    d_out->dvxdt[thread_id] = d_in->dvxdt[thread_id];
    d_out->dvydt[thread_id] = d_in->dvydt[thread_id];

    d_out->u[thread_id] = d_in->u[thread_id];
    d_out->dudt[thread_id] = d_in->dudt[thread_id];

    d_out->h[thread_id] = d_in->h[thread_id];
    d_out->m[thread_id] = d_in->m[thread_id];
    d_out->p[thread_id] = d_in->p[thread_id];
    d_out->omega[thread_id] = d_in->omega[thread_id];
    d_out->temp_timestep[thread_id] = d_in->temp_timestep[thread_id];
    

    double d = 0.0;//d_in->density[thread_id];
    for(int i = 0; i < lattice_size; i++){
        d += (d_in->m[i] * cubic_spline_kernel_periodic(d_in->rx[thread_id], d_in->ry[thread_id], d_in->rx[i], d_in->ry[i], d_in->h[thread_id], d_in->dimensions, d_in->xbound, d_in->ybound));
    }
    
    d_out->density[thread_id] = d;

}


//set each particle's smoothing length and store them to coordinates vector
__global__ void setting_smoothing_length(Coordinates* d_out, Coordinates* d_in){//based on 2.2.2 in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    int lattice_size = d_in->number;

    double f1; // f(h) - equation (2.46) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf
    double f2; // f'(h) - equation (2.48) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf
    
    d_out->number = d_in->number;
    d_out->dimensions = d_in->dimensions;
    d_out->timestep = d_in->timestep;
    d_out->x_min = d_in->x_min;
    d_out->y_min = d_in->y_min;
    d_out->xbound = d_in->xbound;
    d_out->ybound = d_in->ybound;

    d_out->rx[thread_id] = d_in->rx[thread_id];
    d_out->ry[thread_id] = d_in->ry[thread_id];

    d_out->vx[thread_id] = d_in->vx[thread_id];
    d_out->vy[thread_id] = d_in->vy[thread_id];

    d_out->dvxdt[thread_id] = d_in->dvxdt[thread_id];
    d_out->dvydt[thread_id] = d_in->dvydt[thread_id];

    d_out->u[thread_id] = d_in->u[thread_id];
    d_out->dudt[thread_id] = d_in->dudt[thread_id];
    
    d_out->m[thread_id] = d_in->m[thread_id];
    d_out->p[thread_id] = d_in->p[thread_id];
    d_out->temp_timestep[thread_id] = d_in->temp_timestep[thread_id];

    double mass = d_in->m[thread_id];
    double h0 = d_in->h[thread_id];//value of the smoothing length before the first iteration
    double dimensions = d_in->dimensions;
    double omega;
    double density = d_in->density[thread_id];

    f1 = mass * pow(1.2 / h0, dimensions) - density; //η = 1.2 in equation (2.46) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf

    //calculate current Ω
    omega = 1.0;
    for (int i = 0; i < lattice_size; i++){ //first_derivative_kernel was implemented in tools.h
        omega += ((h0 * d_in->m[i] * first_derivative_kernel_periodic(d_in->rx[thread_id], d_in->ry[thread_id], d_in->rx[i], d_in->ry[i], h0, dimensions, d_in->xbound, d_in->ybound)) / (density * dimensions)); //euqation(27) and (28) in Price (2012)
    }

    f2 = (-dimensions) * density * omega / h0; // equation (2.49) in https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis.pdf

    double h_old = h0;
    double h_new = h_old - f1 / f2;

    while((fabs(h_new - h_old) / h0) >= 0.0001){
            
        omega = 0.0;
        density = 0.0;
        for (int i = 0; i < lattice_size; i++){
            density += (d_in->m[i] * cubic_spline_kernel_periodic(d_in->rx[thread_id], d_in->ry[thread_id], d_in->rx[i], d_in->ry[i], h_new, dimensions, d_in->xbound, d_in->ybound)); // calculate density based on new h
            omega += (d_in->m[i] * first_derivative_kernel_periodic(d_in->rx[thread_id], d_in->ry[thread_id], d_in->rx[i], d_in->ry[i], h_new, dimensions, d_in->xbound, d_in->ybound)); // calculate omega based on new h  
        }
        omega = 1.0 + (omega * h_new) / (density * dimensions);
        
        d_out->density[thread_id] = density;
        d_out->omega[thread_id] = omega;
        
        f1 = mass * pow(1.2 / h_new, dimensions) - density;
        f2 = (-dimensions) * density * omega / h_new;
        
        h_old = h_new;
        h_new = h_old - f1 / f2;

    }

    d_out->h[thread_id] = h_old; 

}


//calculate accelerated speed dvdt and dudt at each particle, store them to coordinates vector and return it: equation (44) and (45) in https://www.sciencedirect.com/science/article/pii/S0021999110006753
__global__ void calculate_dvdt_dudt(Coordinates* d_out, Coordinates* d_in){
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    double norm = 10.0 / (7.0 * M_PI); // σ normalisation factor = 10/7π in 2 dimensions here
    int lattice_size = d_in->number;
    int d = d_in->dimensions;
    double x_bound = d_in->xbound;
    double y_bound = d_in->ybound;

    d_out->number = d_in->number;
    d_out->dimensions = d_in->dimensions;
    d_out->timestep = d_in->timestep;
    d_out->x_min = d_in->x_min;
    d_out->y_min = d_in->y_min;
    d_out->xbound = d_in->xbound;
    d_out->ybound = d_in->ybound;

    d_out->rx[thread_id] = d_in->rx[thread_id];
    d_out->ry[thread_id] = d_in->ry[thread_id];

    d_out->vx[thread_id] = d_in->vx[thread_id];
    d_out->vy[thread_id] = d_in->vy[thread_id];

    d_out->u[thread_id] = d_in->u[thread_id];
    
    d_out->h[thread_id] = d_in->h[thread_id];
    d_out->m[thread_id] = d_in->m[thread_id];
    d_out->density[thread_id] = d_in->density[thread_id];
    d_out->omega[thread_id] = d_in->omega[thread_id];
    d_out->temp_timestep[thread_id] = d_in->temp_timestep[thread_id];
    

    double xa = d_in->rx[thread_id]; //x coordinate of the particle this thread deals with
    double ya = d_in->ry[thread_id]; //y coordinate of the particle this thread deals with
    
    double vax = d_in->vx[thread_id]; // velosity of particle this thread deals with in x direction
    double vay = d_in->vy[thread_id]; // velosity of particle this thread deals with in y direction

    double da = d_in->density[thread_id]; //denisty of the particle this thread deals with

    double oa = d_in->omega[thread_id]; //Ω omega of the particle this thread deals with

    double ha = d_in->h[thread_id]; //smoothing length of the particle this thread deals with

    double ua = d_in->u[thread_id]; // internal energy of the particle this thread deals with

    double pa = p(da, ua); //d_out->p[thread_id] = pa; // pressure at the particle this thread deals with

    double ax = 0; //temporary variable ax to store dvdt in x direction at the particle this thread deals with
    double ay = 0; //temporary variable ay to store dvdt in y direction at the particle this thread deals with

    double ut = 0; //temporary variable ut to store dudt at the particle this thread deals with

    for(int j = 0; j < lattice_size; j++){

        if(thread_id != j){

            double xb = d_in->rx[j]; //x coordinate of particle j
            double yb = d_in->ry[j]; //y coordinate of particle j

            double vbx = d_in->vx[j]; // velosity of particle j in x direction
            double vby = d_in->vy[j]; // velosity of particle j in y direction

            double db = d_in->density[j]; //density of particle j

            double ob = d_in->omega[j]; //Ω omega of particle j

            double hb = d_in->h[j]; //smoothing length of particle j

            double ub = d_in->u[j]; // internal energy of particle j
    
            double pb = p(db, ub); // pressure at particle j

            double mb = d_in->m[j]; //mass of particle j
        
            double distance = distance_2d_periodic(xa, ya, xb, yb, x_bound, y_bound);

            double vsig = v_sig(ua, ub, vax, vay, vbx, vby, xa, ya, xb, yb, x_bound, y_bound); 

            double vr = vabrab(vax, vay, vbx, vby, xa, ya, xb, yb, x_bound, y_bound);

            double average_fab = (fab(ha, distance, d) + fab(hb, distance, d)) / 2.0; //(Fab(ha) + Fab(hb)) / 2

            double average_density = (da + db) / 2.0; 

            //calculate dv/dt, du/dt based on equation (44) and (45) in https://www.sciencedirect.com/science/article/pii/S0021999110006753 with considering periodic boundary
            double x_difference = coor_difference(xa, xb, x_bound); // xa - xb in periodic boundary
            double y_difference = coor_difference(ya, yb, y_bound); // ya - yb in periodic boundary


            //calculate dvdt, equation (44) in https://www.sciencedirect.com/science/article/pii/S0021999110006753
            ax -= (mb * x_difference * norm * (pa * f1(distance / ha) / (oa * da * da * pow(ha, d + 1)) + pb * f1(distance / hb) / (ob * db * db * pow(hb, d + 1))) / distance);
            ay -= (mb * y_difference * norm * (pa * f1(distance / ha) / (oa * da * da * pow(ha, d + 1)) + pb * f1(distance / hb) / (ob * db * db * pow(hb, d + 1))) / distance);
        
            //calculate dudt, equation (45) in https://www.sciencedirect.com/science/article/pii/S0021999110006753
            ut += (pa * mb * norm * f1(distance / ha) * ((vax - vbx) * x_difference + (vay - vby) * y_difference) / (distance * pow(ha, d + 1) * oa * da * da));
            
            
            //implement artificial viscosity
            //update dvdt, equation (101) in https://www.sciencedirect.com/science/article/pii/S0021999110006753
            ax += (mb * vsig * vr * average_fab * x_difference / (distance * average_density)); //dv/dt in x direction
            ay += (mb * vsig * vr * average_fab * y_difference / (distance * average_density)); //dv/dt in y direction
        
            //update dudt, equation (104) in https://www.sciencedirect.com/science/article/pii/S0021999110006753
            //Read this please: https://www.cs.mun.ca/~tstricco/papers/Tricco-phdthesis-errata.pdf
            ut -= (mb * average_fab * (0.5 * vsig * vr * vr - sqrt(fabs(pa - pb) / average_density) * (ua - ub)) / average_density);

        }
    
    }

    d_out->p[thread_id] = pa;
    d_out->dvxdt[thread_id] = ax;
    d_out->dvydt[thread_id] = ay;
    d_out->dudt[thread_id] = ut;

}


//update temporary timestep for each particle
__global__ void update_timestep(Coordinates* d_out, Coordinates* d_in){
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    d_out->number = d_in->number;
    d_out->dimensions = d_in->dimensions;
    d_out->timestep = d_in->timestep;
    d_out->x_min = d_in->x_min;
    d_out->y_min = d_in->y_min;
    d_out->xbound = d_in->xbound;
    d_out->ybound = d_in->ybound;

    d_out->rx[thread_id] = d_in->rx[thread_id];
    d_out->ry[thread_id] = d_in->ry[thread_id];

    d_out->vx[thread_id] = d_in->vx[thread_id];
    d_out->vy[thread_id] = d_in->vy[thread_id];

    d_out->dvxdt[thread_id] = d_in->dvxdt[thread_id];
    d_out->dvydt[thread_id] = d_in->dvydt[thread_id];

    d_out->u[thread_id] = d_in->u[thread_id];
    d_out->dudt[thread_id] = d_in->dudt[thread_id];
    
    d_out->h[thread_id] = d_in->h[thread_id];
    d_out->m[thread_id] = d_in->m[thread_id];
    d_out->density[thread_id] = d_in->density[thread_id];
    d_out->p[thread_id] = d_in->p[thread_id];
    d_out->omega[thread_id] = d_in->omega[thread_id];
    
    double cs = cal_cs(d_in->u[thread_id]);
    d_out->temp_timestep[thread_id] = 0.25 * d_in->h[thread_id] / cs;

}


//the first half of leapfrog integration
__global__ void leapfrog_firsthalf(Coordinates* d_out, Coordinates* d_in){
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    double xmin = d_in->x_min;
    double ymin = d_in->y_min;
    double x_bound = d_in->xbound;
    double y_bound = d_in->ybound;

    d_out->number = d_in->number;
    d_out->dimensions = d_in->dimensions;
    d_out->timestep = d_in->timestep;
    d_out->x_min = d_in->x_min;
    d_out->y_min = d_in->y_min;
    d_out->xbound = d_in->xbound;
    d_out->ybound = d_in->ybound;

    d_out->dvxdt[thread_id] = d_in->dvxdt[thread_id];
    d_out->dvydt[thread_id] = d_in->dvydt[thread_id];

    d_out->dudt[thread_id] = d_in->dudt[thread_id];
    
    d_out->h[thread_id] = d_in->h[thread_id];
    d_out->m[thread_id] = d_in->m[thread_id];
    d_out->density[thread_id] = d_in->density[thread_id];
    d_out->p[thread_id] = d_in->p[thread_id];
    d_out->omega[thread_id] = d_in->omega[thread_id];
    d_out->temp_timestep[thread_id] = d_in->temp_timestep[thread_id];

    d_out->u[thread_id] = d_in->u[thread_id] + (d_in->dudt[thread_id] * d_in->timestep / 2.0);
    d_out->vx[thread_id] = d_in->vx[thread_id] + (d_in->dvxdt[thread_id] * d_in->timestep / 2.0);
    d_out->vy[thread_id] = d_in->vy[thread_id] + (d_in->dvydt[thread_id] * d_in->timestep / 2.0);
    d_out->rx[thread_id] = d_in->rx[thread_id] + (d_out->vx[thread_id] * d_in->timestep);
    d_out->ry[thread_id] = d_in->ry[thread_id] + (d_out->vy[thread_id] * d_in->timestep);

    //update x position with considering periodic boundary
    while(d_out->rx[thread_id] < xmin){
        d_out->rx[thread_id] += x_bound;
    }
    while(d_out->rx[thread_id] > xmin + x_bound){
        d_out->rx[thread_id] -= x_bound;
    }

    //update y position with considering periodic boundary
    while(d_out->ry[thread_id] < ymin){
        d_out->ry[thread_id] += y_bound;
    }
    while(d_out->ry[thread_id] > ymin + y_bound){
        d_out->ry[thread_id] -= y_bound;
    }

}


//the second half of leapfrog integration
__global__ void leapfrog_secondhalf(Coordinates* d_out, Coordinates* d_in){
    
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x;

    d_out->number = d_in->number;
    d_out->dimensions = d_in->dimensions;
    d_out->timestep = d_in->timestep;
    d_out->x_min = d_in->x_min;
    d_out->y_min = d_in->y_min;
    d_out->xbound = d_in->xbound;
    d_out->ybound = d_in->ybound;

    d_out->rx[thread_id] = d_in->rx[thread_id];
    d_out->ry[thread_id] = d_in->ry[thread_id];

    d_out->dvxdt[thread_id] = d_in->dvxdt[thread_id];
    d_out->dvydt[thread_id] = d_in->dvydt[thread_id];

    d_out->dudt[thread_id] = d_in->dudt[thread_id];
    
    d_out->h[thread_id] = d_in->h[thread_id];
    d_out->m[thread_id] = d_in->m[thread_id];
    d_out->density[thread_id] = d_in->density[thread_id];
    d_out->p[thread_id] = d_in->p[thread_id];
    d_out->omega[thread_id] = d_in->omega[thread_id];
    d_out->temp_timestep[thread_id] = d_in->temp_timestep[thread_id];

    d_out->u[thread_id] = d_in->u[thread_id] + (d_in->dudt[thread_id] * d_in->timestep / 2.0);
    d_out->vx[thread_id] = d_in->vx[thread_id] + (d_in->dvxdt[thread_id] * d_in->timestep / 2.0);
    d_out->vy[thread_id] = d_in->vy[thread_id] + (d_in->dvydt[thread_id] * d_in->timestep / 2.0);

}


//write particle data to csv
void write_file(){
    
    ofstream myfile;
    myfile.open("particles-cuda.csv");
    myfile << "particle #,x,y,vx,vy,m,Omega,h,density,pressure,dvx/dt,dvy/dt,du/dt\n"; //h: smoothing length
    
    for(int i = 0; i < LATTICE_SIZE; i++){
        myfile << "particle " << i << "," << coordinates.rx[i] << "," << coordinates.ry[i] << "," << coordinates.vx[i] << "," << coordinates.vy[i] << "," << coordinates.m[i] << "," << coordinates.omega[i] << "," << coordinates.h[i] << "," << coordinates.density[i] << "," << coordinates.p[i] << "," << coordinates.dvxdt[i] << "," << coordinates.dvydt[i] << "," << coordinates.dudt[i] << "\n";
    }

    myfile.close();
}


int main(){
    
    coordinates = lattice_init_square(); //create a lattice of particles in 1 * 1 square arrangement
    
    //host memory
    Coordinates* h_in = &coordinates;
    Coordinates* h_out = new Coordinates;

    size_t memory_size = sizeof(coordinates);

    //Declare and allocate device memory
    Coordinates* d_in;
    Coordinates* d_out;
    hipMalloc((void**)&d_in, memory_size); 
    hipMalloc((void**)&d_out, memory_size);

    hipMemcpy(d_in, h_in, memory_size, hipMemcpyHostToDevice);//data transfer from host to device

    calculate_density<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_in);//Kernel function, calculate each particle's density. 4 blocks in total; 625 threads per block.
    hipDeviceSynchronize();
    
    setting_smoothing_length<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//set smoothing length
    hipDeviceSynchronize();

    calculate_dvdt_dudt<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//calculate dv/dt and du/dt
    hipDeviceSynchronize();

    update_timestep<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//update value of timestep for each particle the first time
    hipDeviceSynchronize();

    hipMemcpy(h_out, d_out, memory_size, hipMemcpyDeviceToHost); //data transfer from device to host

    //hipFree(d_in); hipFree(d_out);//free allocated device memory

    coordinates = *h_out; //get the contents of coordinates in host's end


    //implement leapfrog integration, starting here, corresponding to Coordinates leapfrog_integrator(Coordinates coordinates) in week5.cpp
    double min = coordinates.temp_timestep[0]; //temporary variable to store the minimal element in the temp_timestep array
    for(int i = 0; i < LATTICE_SIZE; i++){
        if(coordinates.temp_timestep[i] < min){
            min = coordinates.temp_timestep[i];
        }
    }
    coordinates.timestep = min;


    for(int i = 0; i < 10; i++){

        //host memory
        // h_in = &coordinates;
        // h_out = new Coordinates;

        // memory_size = sizeof(coordinates);

        //allocate device memory
        // hipMalloc((void**)&d_in, memory_size); 
        // hipMalloc((void**)&d_out, memory_size);

        hipMemcpy(d_in, h_in, memory_size, hipMemcpyHostToDevice);//data transfer from host to device
        
        leapfrog_firsthalf<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_in);
        hipDeviceSynchronize();

        calculate_density<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//Kernel function, calculate each particle's density. 4 blocks in total; 625 threads per block.
        hipDeviceSynchronize();
        
        setting_smoothing_length<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//set smoothing length
        hipDeviceSynchronize();

        calculate_dvdt_dudt<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//calculate dv/dt and du/dt
        hipDeviceSynchronize();

        update_timestep<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_out);//update value of timestep for each particle the first time
        hipDeviceSynchronize();

        hipMemcpy(h_out, d_out, memory_size, hipMemcpyDeviceToHost); //data transfer from device to host

        //hipFree(d_in); hipFree(d_out);//free allocated device memory

        coordinates = *h_out; //get the contents of coordinates in host's end


        min = coordinates.temp_timestep[0]; //temporary variable to store the minimal element in the temp_timestep array
        for(int j = 0; j < LATTICE_SIZE; j++){
            if(coordinates.temp_timestep[j] < min){
                min = coordinates.temp_timestep[j];
            }
        }
        coordinates.timestep = min;

        cout << coordinates.timestep << endl;//debug

        //host memory
        // h_in = &coordinates;
        // h_out = new Coordinates;

        // memory_size = sizeof(coordinates);
        
        // //allocate device memory
        // hipMalloc((void**)&d_in, memory_size); 
        // hipMalloc((void**)&d_out, memory_size);

        hipMemcpy(d_in, h_in, memory_size, hipMemcpyHostToDevice);//data transfer from host to device

        leapfrog_secondhalf<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>(d_out, d_in);
        hipDeviceSynchronize();

        hipMemcpy(h_out, d_out, memory_size, hipMemcpyDeviceToHost); //data transfer from device to host

        //hipFree(d_in); hipFree(d_out);//free allocated device memory

        coordinates = *h_out; //get the contents of coordinates in host's end

    }

    hipFree(d_in); hipFree(d_out);//free allocated device memory

    write_file();//write particles' data to a csv file
    
    return 0;
    
}